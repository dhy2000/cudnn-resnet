#include "BatchNorm.h"
#include "timing.h"

BatchNorm::BatchNorm() {}

BatchNorm::BatchNorm(hipdnnHandle_t handle, float* data): handle(handle), input_data(data) {
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&batch_norm_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
}

void BatchNorm::SetScaleAndBias() {
    bn_scale = (float*) malloc(input_c * sizeof(float));
    bn_bias = (float*) malloc(input_c * sizeof(float));
    CUDA_CALL(hipMalloc((void**)&d_bn_scale, input_c * sizeof(float)));
    CUDA_CALL(hipMalloc((void**)&d_bn_bias, input_c * sizeof(float)));

    for(int i = 0; i < input_c; i++) {
        bn_scale[i] = 1;
        bn_bias[i] = 0;
    }

    CUDA_CALL(hipMemcpy(d_bn_scale, bn_scale, input_c * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_bn_bias, bn_bias, input_c * sizeof(float), hipMemcpyHostToDevice));
}

void BatchNorm::SetInputDescriptor(int N, int C, int H, int W) {
    input_n = N;
    input_c = C;
    input_h = H;
    input_w = W;

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                          HIPDNN_TENSOR_NCHW, 
                                          HIPDNN_DATA_FLOAT,
                                          input_n, input_c, input_h, input_w));
    
    #if DEBUG
    printf("Batch Norm Input Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", input_n, input_c, input_h, input_w);
    #endif
}

void BatchNorm::SetBatchNormDescriptor() {
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(batch_norm_descriptor, 
                                          HIPDNN_TENSOR_NCHW, 
                                          HIPDNN_DATA_FLOAT,
                                          1, input_c, 1, 1));
}

void BatchNorm::SetOutputDescriptor() {
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor, 
                                          HIPDNN_TENSOR_NCHW, 
                                          HIPDNN_DATA_FLOAT,
                                          input_n, input_c, input_h, input_w));
    
    #if DEBUG
    printf("Batch Norm Output Shape (NCHW) => N: %d, C: %d, H: %d, W: %d\n", input_n, input_c, input_h, input_w);
    #endif

    CUDA_CALL(hipMalloc(&estimated_mean, input_n * input_c * input_h * input_w * sizeof(float)));
    CUDA_CALL(hipMalloc(&estimated_variance, input_n * input_c * input_h * input_w * sizeof(float)));
    CUDA_CALL(hipMalloc(&output_data, input_n * input_c * input_h * input_w * sizeof(float)));
}

float* BatchNorm::GetOutputData() {
    return output_data;
}

void BatchNorm::Forward() {
    float one = 1;
    float zero = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    starttime(2);
    hipEventRecord(start);
    CUDNN_CALL(hipdnnBatchNormalizationForwardInference(
        handle,
        HIPDNN_BATCHNORM_SPATIAL, /*hipdnnBatchNormMode_t mode*/
        &one,
        &zero,
        input_descriptor,/*const hipdnnTensorDescriptor_t xDesc*/
        input_data,/*const void *x*/
        output_descriptor,/*const hipdnnTensorDescriptor_t yDesc*/
        output_data, /*void *y*/
        batch_norm_descriptor,/*const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc*/
        d_bn_scale,
        d_bn_bias,
        estimated_mean,
        estimated_variance,
        epsilon
    ));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    stoptime(2);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    #if !DEBUG
    printf("%f\n", milliseconds);
    #endif
}

void BatchNorm::Free() {
    CUDA_CALL(hipFree(d_bn_scale));
    CUDA_CALL(hipFree(d_bn_bias));
    CUDA_CALL(hipFree(estimated_mean));
    CUDA_CALL(hipFree(estimated_variance));
    CUDA_CALL(hipFree(input_data));
    free(bn_scale);
    free(bn_bias);

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(batch_norm_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
}
#include "hip/hip_runtime.h"
#include "Utils.h"

__global__ void fill_constant(float *px, float k) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    px[tid] = k; 
}

__global__ void add_identity(float* orig, float* identity, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < size) {
        orig[tid] += identity[tid];
    } 
}

__global__ void copy(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < size) {
        out[tid] = in[tid];
    }
}

void print(const float *data, int n, int c, int h, int w) {
    std::vector<float> buffer(1 << 20);
    CUDA_CALL(hipMemcpy(buffer.data(), data, n * c * h * w * sizeof(float), hipMemcpyDeviceToHost));

    int a = 0;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < c; ++j) {
            std::cout << "n=" << i << ", c=" << j << ":" << std::endl;
            for (int k = 0; k < h; ++k) {
                for (int l = 0; l < w; ++l) {
                    std::cout << std::setw(10) << std::right << buffer[a];
                    ++a;
                }
                std::cout << std::endl;
            }
        }
    }

    std::cout << std::endl;
}
